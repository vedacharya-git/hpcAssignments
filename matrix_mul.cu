#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixMultiply(float *A, float *B, float *C, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        float sum = 0.0;
        for (int i = 0; i < colsA; i++) {
            sum += A[row * colsA + i] * B[i * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

int main() {
    int rowsA = 3, colsA = 3;
    int rowsB = 3, colsB = 3;
    int rowsC = rowsA, colsC = colsB;
    int sizeA = rowsA * colsA * sizeof(float);
    int sizeB = rowsB * colsB * sizeof(float);
    int sizeC = rowsC * colsC * sizeof(float);

    float h_A[] = {1, 2, 3,
                4, 5, 6,
                7, 8, 9};
    float h_B[] = {9, 8, 7,
                6, 5, 4,
                3, 2, 1};
    float h_C[9];

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeA);
    hipMalloc((void **)&d_B, sizeB);
    hipMalloc((void **)&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((colsC + blockDim.x - 1) / blockDim.x,
                (rowsC + blockDim.y - 1) / blockDim.y);

    matrixMultiply<<<gridDim, blockDim>>>(d_A, d_B, d_C, rowsA, colsA, colsB);

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    printf("Resulting matrix C:\n");
    for (int i = 0; i < rowsC; i++) {
        for (int j = 0; j < colsC; j++) {
            printf("%.2f ", h_C[i * colsC + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

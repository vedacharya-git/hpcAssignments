#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixAdd(float *A, float *B, float *C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int idx = row * cols + col;

    if (row < rows && col < cols) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int rows = 4;
    int cols = 4;
    int size = rows * cols * sizeof(float);

    float h_A[16] = {1, 2, 3, 4,
                    5, 6, 7, 8,
                    9, 10, 11, 12,
                    13, 14, 15, 16};
    float h_B[16] = {16, 15, 14, 13,
                    12, 11, 10, 9,
                    8, 7, 6, 5,
                    4, 3, 2, 1};
    float h_C[16];

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 blockDim(2, 2); // Block of 2x2 threads
    dim3 gridDim((cols + blockDim.x - 1) / blockDim.x,
                (rows + blockDim.y - 1) / blockDim.y);

    matrixAdd<<<gridDim, blockDim>>>(d_A, d_B, d_C, rows, cols);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Resulting matrix C:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%.2f ", h_C[i * cols + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
